#include "hip/hip_runtime.h"

//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#include <library_daq.h>

// CUDA = Computer Device Unified Architecture

__global__ void kernel_correct_times(unsigned int *ct);




//
// main code
//

int main(int argc, const char **argv)
{


  /////////////////////
  // initialise card //
  /////////////////////
  findCudaDevice(argc, argv);


  // initialise CUDA timing
  bool use_timing = true;
  if( use_timing ){
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }
  hipEventCreate(&total_start);
  hipEventCreate(&total_stop);
  float elapsed_parameters = 0, elapsed_pmts = 0, elapsed_detector = 0, elapsed_verteces = 0,
    elapsed_threads = 0, elapsed_tof = 0, elapsed_memory_tofs_dev = 0, elapsed_memory_candidates_host = 0, elapsed_tofs_copy_dev = 0,
    elapsed_input = 0, elapsed_memory_dev = 0, elapsed_copy_dev = 0, elapsed_kernel = 0, 
    elapsed_threads_candidates = 0, elapsed_candidates_memory_dev = 0, elapsed_candidates_kernel = 0,
    elapsed_candidates_copy_host = 0, choose_candidates = 0, elapsed_coalesce = 0, elapsed_gates = 0, elapsed_free = 0, elapsed_total = 0,
    elapsed_tofs_free = 0, elapsed_reset = 0, elapsed_write_output = 0;
  use_verbose = true;


  ////////////////////
  // inspect device //
  ////////////////////
  // set: max_n_threads_per_block, max_n_blocks
  print_gpu_properties();




  ////////////////
  // read PMTs  //
  ////////////////
  // set: n_PMTs, PMT_x, PMT_y, PMT_z
  if( use_timing )
    start_c_clock();
  event_file_base = "all_hits_";
  event_file_suffix = ".txt";
  detector_file = "detector.txt";
  pmts_file = "all_pmts.txt";
  output_file_base = "all_hits_emerald_threshold_";
  if( !read_the_pmts() ) return 0;
  if( use_timing )
    elapsed_pmts = stop_c_clock();


  ///////////////////////
  // define parameters //
  ///////////////////////
  if( use_timing )
    start_c_clock();
  read_user_parameters();
  set_output_file();
  if( use_verbose ){
    printf(" --- user parameters \n");
    printf(" distance between test verteces = %f cm \n", distance_between_verteces);
    printf(" time step size = %d ns \n", time_step_size);
    printf(" threshold_number_of_pmts = %d \n", threshold_number_of_pmts);
    printf(" coalesce_time = %f ns \n", coalesce_time);
    printf(" trigger_gate_up = %f ns \n", trigger_gate_up);
    printf(" trigger_gate_down = %f ns \n", trigger_gate_down);
  }
  if( use_timing )
    elapsed_parameters = stop_c_clock();




  /////////////////////
  // read detector ////
  /////////////////////
  // set: detector_height, detector_radius, pmt_radius
  if( use_timing )
    start_c_clock();
  if( !read_the_detector() ) return 0;
  if( use_timing )
    elapsed_detector = stop_c_clock();




  ////////////////////////
  // make test verteces //
  ////////////////////////
  // set: n_test_verteces, vertex_x, vertex_y, vertex_z
  // use: detector_height, detector_radius
  if( use_timing )
    start_c_clock();
  make_test_verteces();
  if( use_timing )
    elapsed_verteces = stop_c_clock();



  //////////////////////////////
  // table of times_of_flight //
  //////////////////////////////
  // set: host_times_of_flight, time_offset
  // use: n_test_verteces, vertex_x, vertex_y, vertex_z, n_PMTs, PMT_x, PMT_y, PMT_z
  // malloc: host_times_of_flight
  if( use_timing )
    start_c_clock();
  make_table_of_tofs();
  if( use_timing )
    elapsed_tof = stop_c_clock();



  ////////////////////////////////////
  // allocate tofs memory on device //
  ////////////////////////////////////
  // use: n_test_verteces, n_PMTs
  // cudamalloc: device_times_of_flight
  if( use_timing )
    start_cuda_clock();
  allocate_tofs_memory_on_device();
  if( use_timing )
    elapsed_memory_tofs_dev = stop_cuda_clock();


  ////////////////////////////////
  // fill tofs memory on device //
  ////////////////////////////////
  // use: n_test_verteces, n_PMTs
  // memcpy: device_times_of_flight, constant_time_step_size, constant_n_test_verteces, constant_n_PMTs
  // texture: tex_times_of_flight
  if( use_timing )
    start_cuda_clock();
  fill_tofs_memory_on_device();
  if( use_timing )
    elapsed_tofs_copy_dev = stop_cuda_clock();


  ///////////////////////
  // initialize output //
  ///////////////////////
  initialize_output();



  start_total_cuda_clock();

  int n_events = 0;

  while( set_input_file_for_event(n_events) ){

    printf(" ------ analyzing event %d \n", n_events+1);

    ////////////////
    // read input //
    ////////////////
    // set: n_hits, host_ids, host_times, time_offset, n_time_bins
    // use: time_offset, n_test_verteces
    // memcpy: constant_n_time_bins, constant_n_hits
    if( use_timing )
      start_c_clock();
    if( !read_the_input() ) return 0;
    if( use_timing )
      elapsed_input += stop_c_clock();
  


    ////////////////////////////////////////
    // allocate candidates memory on host //
    ////////////////////////////////////////
    // use: n_time_bins, n_hits
    // malloc: host_max_number_of_pmts_in_time_bin, host_vertex_with_max_n_pmts
    if( use_timing )
      start_cuda_clock();
    allocate_candidates_memory_on_host();
    if( use_timing )
      elapsed_memory_candidates_host += stop_cuda_clock();


    ////////////////////////////////////////////////
    // set number of blocks and threads per block //
    ////////////////////////////////////////////////
    // set: number_of_kernel_blocks, number_of_threads_per_block
    // use: n_test_verteces, n_hits
    if( use_timing )
      start_c_clock();
    if( !setup_threads_for_tof_2d() ) return 0;
    if( use_timing )
      elapsed_threads += stop_c_clock();



    ///////////////////////////////////////
    // allocate correct memory on device //
    ///////////////////////////////////////
    // use: n_test_verteces, n_hits, n_time_bins
    // cudamalloc: device_ids, device_times, device_n_pmts_per_time_bin
    if( use_timing )
      start_cuda_clock();
    allocate_correct_memory_on_device();
    if( use_timing )
      elapsed_memory_dev += stop_cuda_clock();


    //////////////////////////////////////
    // copy input into device variables //
    //////////////////////////////////////
    // use: n_hits
    // memcpy: device_ids, device_times, constant_time_offset
    // texture: tex_ids, tex_times
    if( use_timing )
      start_cuda_clock();
    fill_correct_memory_on_device();
    if( use_timing )
      elapsed_copy_dev += stop_cuda_clock();



    ////////////////////
    // execute kernel //
    ////////////////////
    if( use_timing )
      start_cuda_clock();
    printf(" --- execute kernel \n");
    kernel_correct_times<<<number_of_kernel_blocks_3d,number_of_threads_per_block_3d>>>(device_n_pmts_per_time_bin);
    getLastCudaError("correct_kernel execution failed\n");
    if( use_timing )
      elapsed_kernel += stop_cuda_clock();



    //////////////////////////////////
    // setup threads for candidates //
    //////////////////////////////////
    // set: number_of_kernel_blocks, number_of_threads_per_block
    // use: n_time_bins
    if( use_timing )
      start_c_clock();
    if( !setup_threads_to_find_candidates() ) return 0;
    if( use_timing )
      elapsed_threads_candidates += stop_c_clock();



    //////////////////////////////////////////
    // allocate candidates memory on device //
    //////////////////////////////////////////
    // use: n_time_bins
    // cudamalloc: device_max_number_of_pmts_in_time_bin, device_vertex_with_max_n_pmts
    if( use_timing )
      start_cuda_clock();
    allocate_candidates_memory_on_device();
    if( use_timing )
      elapsed_candidates_memory_dev += stop_cuda_clock();



    /////////////////////////////////////
    // find candidates above threshold //
    /////////////////////////////////////
    if( use_timing )
      start_cuda_clock();
    if( use_verbose )
      printf(" --- execute candidates kernel \n");
    kernel_find_vertex_with_max_npmts_in_timebin<<<number_of_kernel_blocks,number_of_threads_per_block>>>(device_n_pmts_per_time_bin, device_max_number_of_pmts_in_time_bin, device_vertex_with_max_n_pmts);
    getLastCudaError("candidates_kernel execution failed\n");
    if( use_timing )
      elapsed_candidates_kernel += stop_cuda_clock();




    /////////////////////////////////////////
    // copy candidates from device to host //
    /////////////////////////////////////////
    // use: n_time_bins
    // memcpy: host_max_number_of_pmts_in_time_bin, host_vertex_with_max_n_pmts
    if( use_timing )
      start_cuda_clock();
    if( use_verbose )
      printf(" --- copy candidates from device to host \n");
    copy_candidates_from_device_to_host();
    if( use_timing )
      elapsed_candidates_copy_host += stop_cuda_clock();



    ///////////////////////////////////////
    // choose candidates above threshold //
    ///////////////////////////////////////
    if( use_timing )
      start_cuda_clock();
    if( use_verbose )
      printf(" --- choose candidates above threshold \n");
    choose_candidates_above_threshold();
    if( use_timing )
      choose_candidates = stop_cuda_clock();



    ///////////////////////
    // coalesce triggers //
    ///////////////////////
    if( use_timing )
      start_cuda_clock();
    coalesce_triggers();
    if( use_timing )
      elapsed_coalesce += stop_cuda_clock();




    //////////////////////////////////
    // separate triggers into gates //
    //////////////////////////////////
    if( use_timing )
      start_cuda_clock();
    separate_triggers_into_gates();
    if( use_timing )
      elapsed_gates += stop_cuda_clock();



    //////////////////
    // write output //
    //////////////////
    if( use_timing )
      start_cuda_clock();
    write_output();
    if( use_timing )
      elapsed_write_output += stop_cuda_clock();

    /////////////////////////////
    // deallocate event memory //
    /////////////////////////////
    if( use_timing )
      start_cuda_clock();
    if( use_verbose )
      printf(" --- deallocate memory \n");
    free_event_memories();
    if( use_timing )
      elapsed_free += stop_cuda_clock();

    n_events ++;

  }

  elapsed_total += stop_total_cuda_clock();


  printf(" ------ analyzed %d events \n", n_events);

  ///////////////////////
  // normalize timings //
  ///////////////////////
  if( use_timing ){
    elapsed_input /= n_events;
    elapsed_memory_candidates_host /= n_events;
    elapsed_threads /= n_events;
    elapsed_memory_dev /= n_events;
    elapsed_copy_dev /= n_events;
    elapsed_kernel /= n_events;
    elapsed_threads_candidates /= n_events;
    elapsed_candidates_memory_dev /= n_events;
    elapsed_candidates_kernel /= n_events;
    elapsed_candidates_copy_host /= n_events;
    elapsed_coalesce /= n_events;
    elapsed_gates /= n_events;
    elapsed_write_output /= n_events;
    elapsed_free /= n_events;
  }
  elapsed_total /= n_events;



  //////////////////////////////
  // deallocate global memory //
  //////////////////////////////
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- deallocate tofs memory \n");
  free_global_memories();
  if( use_timing )
    elapsed_tofs_free = stop_cuda_clock();



  //////////////////
  // reset device //
  //////////////////
  // -- needed to flush the buffer which holds printf from each thread
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- reset device \n");
  //  hipDeviceReset();
  if( use_timing )
    elapsed_reset = stop_cuda_clock();



  //////////////////
  // print timing //
  //////////////////
  if( use_timing ){
    printf(" user parameters time : %f ms \n", elapsed_parameters);
    printf(" read pmts execution time : %f ms \n", elapsed_pmts);
    printf(" read detector execution time : %f ms \n", elapsed_detector);
    printf(" make test verteces execution time : %f ms \n", elapsed_verteces);
    printf(" setup threads candidates execution time : %f ms \n", elapsed_threads_candidates);
    printf(" make table of tofs execution time : %f ms \n", elapsed_tof);
    printf(" allocate tofs memory on device execution time : %f ms \n", elapsed_memory_tofs_dev);
    printf(" fill tofs memory on device execution time : %f ms \n", elapsed_tofs_copy_dev);
    printf(" deallocate tofs memory execution time : %f ms \n", elapsed_tofs_free);
    printf(" device reset execution time : %f ms \n", elapsed_reset);
    printf(" read input execution time : %f ms (%f) \n", elapsed_input, elapsed_input/elapsed_total);
    printf(" allocate candidates memory on host execution time : %f ms (%f) \n", elapsed_memory_candidates_host, elapsed_memory_candidates_host/elapsed_total);
    printf(" setup threads execution time : %f ms (%f) \n", elapsed_threads, elapsed_threads/elapsed_total);
    printf(" allocate memory on device execution time : %f ms (%f) \n", elapsed_memory_dev, elapsed_memory_dev/elapsed_total);
    printf(" fill memory on device execution time : %f ms (%f) \n", elapsed_copy_dev, elapsed_copy_dev/elapsed_total);
    printf(" correct kernel execution time : %f ms (%f) \n", elapsed_kernel, elapsed_kernel/elapsed_total);
    printf(" allocate candidates memory on device execution time : %f ms (%f) \n", elapsed_candidates_memory_dev, elapsed_candidates_memory_dev/elapsed_total);
    printf(" copy candidates to host execution time : %f ms (%f) \n", elapsed_candidates_copy_host, elapsed_candidates_copy_host/elapsed_total);
    printf(" choose candidates execution time : %f ms (%f) \n", choose_candidates, choose_candidates/elapsed_total);
    printf(" candidates kernel execution time : %f ms (%f) \n", elapsed_candidates_kernel, elapsed_candidates_kernel/elapsed_total);
    printf(" coalesce triggers execution time : %f ms (%f) \n", elapsed_coalesce, elapsed_coalesce/elapsed_total);
    printf(" separate triggers into gates execution time : %f ms (%f) \n", elapsed_gates, elapsed_gates/elapsed_total);
    printf(" write output execution time : %f ms (%f) \n", elapsed_write_output, elapsed_write_output/elapsed_total);
    printf(" deallocate memory execution time : %f ms (%f) \n", elapsed_free, elapsed_free/elapsed_total);
  }
  printf(" total execution time : %f ms \n", elapsed_total);


  return 1;
}



//
// kernel routine
// 

// __global__ identifier says it's a kernel function
__global__ void kernel_correct_times(unsigned int *ct){

  // get unique id for each thread in each block
  unsigned int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
  unsigned int tid_y = threadIdx.y + blockDim.y*blockIdx.y;

  // map the two 2D indices to a single linear, 1D index
  int tid = tid_y * gridDim.x * blockDim.x + tid_x;

  // tid runs from 0 to n_test_verteces * n_hits:
  //      vertex 0           vertex 1       ...     vertex m
  // (hit 0, ..., hit n; hit 0, ..., hit n; ...; hit 0, ..., hit n);

  unsigned int vertex_index = (int)(tid/constant_n_hits);
  unsigned int hit_index = tid % constant_n_hits;

  //  printf(" tid %d tidx %d tidy %d v %d h %d \n", tid, tid_x, tid_y, vertex_index, hit_index);

  //    printf( " threadi %d blockdim %d blockid %d, tid %d, vertex_index %d, hit %d \n",
  //  	  threadIdx.x, blockDim.x, blockIdx.x, tid,
  //	  vertex_index, hit_index);

  // skip if thread is assigned to nonexistent vertex
  if( vertex_index >= constant_n_test_verteces ) return;

  // skip if thread is assigned to nonexistent hit
  if( hit_index >= constant_n_hits ) return;

  unsigned int vertex_block = constant_n_time_bins*vertex_index;

  unsigned int vertex_block2 = constant_n_PMTs*vertex_index;

  atomicAdd(&
	    ct[
	       device_get_time_index(
				     int(floor(
					       (tex1Dfetch(tex_times,hit_index)
						- tex1Dfetch(tex_times_of_flight,
							     device_get_distance_index(
										       tex1Dfetch(tex_ids,hit_index),
										       vertex_block2
										       )
							     )
						+ constant_time_offset)/constant_time_step_size
					       )
					 ),
				     vertex_block
				     )
	       ]
	    ,1);

  //  printf( " hit %d (nh %d) id %d t %d; vertex %d (nv %d) tof %f  %d \n", hit_index, constant_n_hits, ids[hit_index], t[hit_index], vertex_index, constant_n_test_verteces, tof, ct[time_index]);

  return;

}


